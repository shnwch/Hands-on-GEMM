
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <iostream>
constexpr size_t BLOCK_SIZE = 16; // we assume that every block has equal blockDim.x and blockDim.y
constexpr size_t BLOCK_M = 128;   // These const values decide how many thing a thread compute and the amount of shared memory to allocate.
constexpr size_t BLOCK_N = 128;
constexpr size_t BLOCK_K = 8; // don't set 64 here, it will cause bank conflict and lower occupancy.
constexpr size_t BLOCK_M_COMPUTE = BLOCK_M / BLOCK_SIZE; // Mthread 8 = 128 / 16
constexpr size_t BLOCK_N_COMPUTE = BLOCK_N / BLOCK_SIZE;  // Nthread

constexpr int shared_memory_A = BLOCK_M * BLOCK_K;
constexpr int shared_memory_B = BLOCK_N * BLOCK_K;
constexpr int shared_memory_element = shared_memory_A + shared_memory_B;
constexpr int shared_memory_size = shared_memory_element * sizeof(float); // shared memory to use.
#define colM(a, i, j, lda) a[((j) * (lda)) + (i)]
#define rowM(a, i, j, lda) a[(j) + (i) * (lda)]

__global__ void matrixMul(const float *A, const float *B, float *C,
                          int M, int N, int K, float alpha, float beta)
{
    const size_t baseX = blockIdx.x * blockDim.x * BLOCK_M_COMPUTE; 
    const size_t baseY = blockIdx.y * blockDim.y * BLOCK_N_COMPUTE;

    const int moveNum = shared_memory_element / (BLOCK_SIZE * BLOCK_SIZE) / 2; 
    const size_t baseIdx = threadIdx.y * blockDim.y + threadIdx.x; //应该是乘blockDim.x？？？？

    constexpr size_t threadsNum = BLOCK_SIZE * BLOCK_SIZE;

    float c[BLOCK_M_COMPUTE * BLOCK_N_COMPUTE] = {};
    float resC[BLOCK_M_COMPUTE * BLOCK_N_COMPUTE] = {};

    __shared__ float subA[BLOCK_M * BLOCK_K];
    __shared__ float subB[BLOCK_N * BLOCK_K];

    float4 regB[BLOCK_M_COMPUTE / 4]; // [8/4] float4  hopefully, these should reside in register.
    float4 regA[BLOCK_M_COMPUTE / 4];

    const float *baseA = A + baseY * K;
    const float *baseB = B + baseX;
    float *baseC = C + (baseY + threadIdx.x * BLOCK_M_COMPUTE) * N + baseX + threadIdx.y * BLOCK_N_COMPUTE;

    int rowA = baseIdx / 2, rowB = baseIdx / (BLOCK_N / 4), colA = (baseIdx & 1) * 4, colB = (baseIdx * 4) % BLOCK_N;
    // rowA = baseIdx / 2 是因为每两行thread 读取一行K大小的A   注意 这里是用来读取的 不是用来定位C的

    for (int i = 0; i < K; i += BLOCK_K)
    {
        regB[0] = *reinterpret_cast<const float4 *>(baseB + i * N + rowB * N + colB); //用寄存器从global读取 当个中转站
        regA[0] = *reinterpret_cast<const float4 *>(baseA + i + rowA * K + colA);
        *reinterpret_cast<float4 *>(&subB[baseIdx * 4]) = regB[0];  //一个线程只负责读取A B各4个数  猪脑子 妈的
        subA[rowA + colA * BLOCK_M] = regA[0].x;
        subA[rowA + (colA + 1) * BLOCK_M] = regA[0].y;
        subA[rowA + (colA + 2) * BLOCK_M] = regA[0].z;
        subA[rowA + (colA + 3) * BLOCK_M] = regA[0].w;

        __syncthreads();
#pragma unroll
        for (int ii = 0; ii < BLOCK_K; ii++)
        {
            regA[0] = *reinterpret_cast<float4 *>(&subA[(threadIdx.x * BLOCK_M_COMPUTE) + ii * BLOCK_M]);
            regA[1] = *reinterpret_cast<float4 *>(&subA[(threadIdx.x * BLOCK_M_COMPUTE + 4) + ii * BLOCK_M]);

            regB[0] = *reinterpret_cast<float4 *>(&subB[threadIdx.y * BLOCK_N_COMPUTE + BLOCK_N * ii]);
            regB[1] = *reinterpret_cast<float4 *>(&subB[threadIdx.y * BLOCK_N_COMPUTE + 4 + BLOCK_N * ii]);

#pragma unroll
            for (int cpi = 0; cpi < BLOCK_M_COMPUTE / 4; cpi++)
            {
#pragma unroll
                for (int cpj = 0; cpj < BLOCK_N_COMPUTE / 4; cpj++)
                {
                    c[cpi * 4 * BLOCK_M_COMPUTE + cpj * 4] += regA[cpi].x * regB[cpj].x;
                    c[cpi * 4 * BLOCK_M_COMPUTE + cpj * 4 + 1] += regA[cpi].x * regB[cpj].y;
                    c[cpi * 4 * BLOCK_M_COMPUTE + cpj * 4 + 2] += regA[cpi].x * regB[cpj].z;
                    c[cpi * 4 * BLOCK_M_COMPUTE + cpj * 4 + 3] += regA[cpi].x * regB[cpj].w;

                    c[(cpi * 4 + 1) * BLOCK_M_COMPUTE + cpj * 4] += regA[cpi].y * regB[cpj].x;
                    c[(cpi * 4 + 1) * BLOCK_M_COMPUTE + cpj * 4 + 1] += regA[cpi].y * regB[cpj].y;
                    c[(cpi * 4 + 1) * BLOCK_M_COMPUTE + cpj * 4 + 2] += regA[cpi].y * regB[cpj].z;
                    c[(cpi * 4 + 1) * BLOCK_M_COMPUTE + cpj * 4 + 3] += regA[cpi].y * regB[cpj].w;

                    c[(cpi * 4 + 2) * BLOCK_M_COMPUTE + cpj * 4] += regA[cpi].z * regB[cpj].x;
                    c[(cpi * 4 + 2) * BLOCK_M_COMPUTE + cpj * 4 + 1] += regA[cpi].z * regB[cpj].y;
                    c[(cpi * 4 + 2) * BLOCK_M_COMPUTE + cpj * 4 + 2] += regA[cpi].z * regB[cpj].z;
                    c[(cpi * 4 + 2) * BLOCK_M_COMPUTE + cpj * 4 + 3] += regA[cpi].z * regB[cpj].w;

                    c[(cpi * 4 + 3) * BLOCK_M_COMPUTE + cpj * 4] += regA[cpi].w * regB[cpj].x;
                    c[(cpi * 4 + 3) * BLOCK_M_COMPUTE + cpj * 4 + 1] += regA[cpi].w * regB[cpj].y;
                    c[(cpi * 4 + 3) * BLOCK_M_COMPUTE + cpj * 4 + 2] += regA[cpi].w * regB[cpj].z;
                    c[(cpi * 4 + 3) * BLOCK_M_COMPUTE + cpj * 4 + 3] += regA[cpi].w * regB[cpj].w;
                }
            }
        }
        __syncthreads();
    }

#pragma unroll
    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
#pragma unroll
        for (int j = 0; j < BLOCK_N_COMPUTE; j += 4)
            *reinterpret_cast<float4 *>(&resC[i * BLOCK_M_COMPUTE + j]) = *reinterpret_cast<float4 *>(&baseC[i * N + j]);

#pragma unroll
    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
#pragma unroll
        for (int j = 0; j < BLOCK_N_COMPUTE; j++)
            resC[i * BLOCK_M_COMPUTE + j] = resC[i * BLOCK_M_COMPUTE + j] * beta + alpha * c[i * BLOCK_M_COMPUTE + j];

#pragma unroll
    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
#pragma unroll
        for (int j = 0; j < BLOCK_N_COMPUTE; j += 4)
            *reinterpret_cast<float4 *>(&baseC[i * N + j]) = *reinterpret_cast<float4 *>(&resC[i * BLOCK_M_COMPUTE + j]);
}

void sgemm(int M, int N, int K, float *a, float *b, float *c, float alpha = 1, float beta = 0)
{
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((M + BLOCK_M - 1) / BLOCK_M, (N + BLOCK_N - 1) / BLOCK_N);
#ifdef __HIPCC__ // workaround for stupid vscode intellisense
    matrixMul<<<numBlocks, threadsPerBlock>>>(a, b, c, M, N, K, alpha, beta);
#endif
}