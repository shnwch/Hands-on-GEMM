#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#ifndef __CUDACC__
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
void __syncthreads(); // workaround __syncthreads warning
#endif
#include <iostream>
const size_t BLOCK_SIZE = 16; // we assume that every block has equal blockDim.x and blockDim.y
const size_t BLOCK_M = 128;   // These const values decide how many thing a thread compute and the amount of shared memory to allocate.
const size_t BLOCK_N = 128;
const size_t BLOCK_K = 8; // don't set 64 here, it will cause bank conflict and lower occupancy.
const size_t BLOCK_M_COMPUTE = BLOCK_M / BLOCK_SIZE;
const size_t BLOCK_N_COMPUTE = BLOCK_N / BLOCK_SIZE;

const int shared_memory_A = BLOCK_M * BLOCK_K;
const int shared_memory_B = BLOCK_N * BLOCK_K;
const int shared_memory_element = shared_memory_A + shared_memory_B;
const int shared_memory_size = shared_memory_element * sizeof(float); // shared memory to use.
#define colM(a, i, j, lda) a[((j) * (lda)) + (i)]
#define rowM(a, i, j, lda) a[(j) + (i) * (lda)]

__forceinline__ __device__ float convertColIdx(int idx, const float *begin, int subM, int subN, int N)
{
    int m = idx / subM, n = idx % subM;
    return begin[m + n * N];
}

__forceinline__ __device__ float convertRowIdx(int idx, const float *begin, int subM, int subN, int N)
{
    int m = idx / subN, n = idx % subN;
    return begin[m * N + n];
}

__global__ void matrixMul(const float *A, const float *B, float *C,
                          int M, int N, int K, float alpha, float beta)
{
    const size_t baseX = blockIdx.x * blockDim.x * BLOCK_M_COMPUTE;
    const size_t baseY = blockIdx.y * blockDim.y * BLOCK_N_COMPUTE;

    const int moveNum = shared_memory_element / (BLOCK_SIZE * BLOCK_SIZE) / 2;
    const size_t baseIdx = threadIdx.x * blockDim.x + threadIdx.y;

    float c[BLOCK_M_COMPUTE * BLOCK_N_COMPUTE] = {};

    __shared__ float subA[BLOCK_M * BLOCK_K];
    __shared__ float subB[BLOCK_N * BLOCK_K];

    float regB[BLOCK_M_COMPUTE]; // hopefully, these should reside in register.
    float regA;

    for (int i = 0; i < K; i += BLOCK_K)
    {
#pragma unroll
        for (int idx = 0; idx < moveNum; idx++)
        {
            subA[baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE] = convertColIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, A + baseX * K + i, BLOCK_M, BLOCK_K, K);
            subB[baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE] = convertRowIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, B + baseY + i * N, BLOCK_K, BLOCK_N, N);
        }
        __syncthreads();
#pragma unroll(4)
        for (int ii = 0; ii < BLOCK_K; ii++)
        {
#pragma unroll
            for (int cpj = 0; cpj < BLOCK_N_COMPUTE; cpj++)
            {
                regB[cpj] = subB[threadIdx.y * BLOCK_N_COMPUTE + cpj + BLOCK_N * ii];
            }
#pragma unroll
            for (int cpi = 0; cpi < BLOCK_M_COMPUTE; cpi++)
            {
                regA = subA[(threadIdx.x * BLOCK_M_COMPUTE + cpi) + ii * BLOCK_M];
#pragma unroll
                for (int cpj = 0; cpj < BLOCK_N_COMPUTE; cpj++)
                {
                    c[cpi * BLOCK_M_COMPUTE + cpj] += regA * regB[cpj];
                }
            }
        }
        __syncthreads();
    }

#pragma unroll
    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
#pragma unroll
        for (int j = 0; j < BLOCK_N_COMPUTE; j++)
            C[(baseX + threadIdx.x * BLOCK_M_COMPUTE + i) * N + baseY + threadIdx.y * BLOCK_N_COMPUTE + j] = beta * C[(baseX + threadIdx.x * BLOCK_M_COMPUTE + i) * N + baseY + threadIdx.y * BLOCK_N_COMPUTE + j] + alpha * c[i * BLOCK_M_COMPUTE + j];
}

void sgemm(int M, int N, int K, float *a, float *b, float *c, float alpha = 1, float beta = 0)
{
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((M + BLOCK_M - 1) / BLOCK_M, (N + BLOCK_N - 1) / BLOCK_N);
#ifdef __HIPCC__ // workaround for stupid vscode intellisense
    matrixMul<<<numBlocks, threadsPerBlock>>>(a, b, c, M, N, K, alpha, beta);
#endif
}
