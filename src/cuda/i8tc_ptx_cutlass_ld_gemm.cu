#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <mma.h>
using namespace nvcuda;
constexpr int WMMA_M = 16;
constexpr int WMMA_N = 16;
constexpr int WMMA_K = 16;
constexpr int BLOCK_M = 256;
constexpr int BLOCK_N = 128;
constexpr int BLOCK_K = 64;
#include <iostream>

__global__ void i8gemm256x128x64(const int8_t *A, const int8_t *B, int32_t *C,
                                 int M, int N, int K, const int32_t alpha, const int32_t beta)
{
    const int lda = K;
    const int ldb = K;
    const int ldc = N;

    constexpr int sharedLda = 16;
    constexpr int sharedLdb = 16;

    const size_t baseIdx = threadIdx.x;

    const auto warpM = (baseIdx / 32) / 4;
    const auto warpN = (baseIdx / 32) % 4;
    const auto laneId = baseIdx % 32;
    const auto warpId = baseIdx / 32;

    const auto baseA = A + blockIdx.x * BLOCK_M * lda;
    const auto baseB = B + blockIdx.y * BLOCK_N * ldb;
    const auto baseC = C + blockIdx.x * BLOCK_M * ldc + blockIdx.y * BLOCK_N + (warpId / 2) * 64 * ldc + (warpId & 1) * 64;

    constexpr auto sharedASize = BLOCK_M * BLOCK_K;
    constexpr auto sharedBSize = BLOCK_N * BLOCK_K;

    __shared__ int8_t shared_mem[sharedASize + sharedBSize];
    auto sharedA = shared_mem;
    auto sharedB = shared_mem + sharedASize;

    int32_t frag_c[64][2] = {}; // Initialize to 0.
    int32_t frag_a[8][4], frag_b[8][4];

    using copy_t = float4; // I use float to store int, haha.

    copy_t preA[4], preB[2];

#pragma unroll
    for (int k = 0; k < K; k += BLOCK_K)
    {
        // Do 64x64x64 (mnk) mma at a time according to cutlass.
#pragma unroll
        for (int i = 0; i < 4; i++)
        {
            preA[i] = *reinterpret_cast<const copy_t *>(&baseA[(baseIdx % 8 + warpId * 8 + i * 64) * lda + (laneId / 8) * 16 + k]);
        }

#pragma unroll
        for (int i = 0; i < 2; i++)
        {
            preB[i] = *reinterpret_cast<const copy_t *>(&baseB[(baseIdx % 8 + warpId * 8 + i * 64) * ldb + (laneId / 8) * 16 + k]);
        }

#pragma unroll
        for (int i = 0; i < 4; i++)
        {
            *reinterpret_cast<copy_t *>(&sharedA[(baseIdx % 8 + warpId * 8 + i * 64 + (laneId / 8) * BLOCK_M) * sharedLda]) = preA[i];
        }

// Need transpose here, I leave it here for now.
#pragma unroll
        for (int i = 0; i < 2; i++)
        {
            *reinterpret_cast<copy_t *>(&sharedB[(baseIdx % 8 + warpId * 8 + i * 64 + (laneId / 8) * BLOCK_N) * sharedLdb]) = preB[i];
        }

        __syncthreads();
        // Load matrix in 4 stages, could try warp shuff and overlap in the future.
        for (int i = 0; i < 8; i++)
        {
            auto ldA = __cvta_generic_to_shared(&sharedA[((warpId / 2) * 64 + i * 8 + laneId % 8 + (laneId / 8) * BLOCK_M) * sharedLda]);
            auto ldB = __cvta_generic_to_shared(&sharedB[((warpId % 2) * 64 + i * 8 + laneId % 8 + (laneId / 8) * BLOCK_N) * sharedLdb]);
            asm volatile(
                "ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%8];"
                "ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%4, %5, %6, %7}, [%9];"
                : "=r"(frag_a[i][0]), "=r"(frag_a[i][1]), "=r"(frag_a[i][2]), "=r"(frag_a[i][3]), "=r"(frag_b[i][0]), "=r"(frag_b[i][1]), "=r"(frag_b[i][2]), "=r"(frag_b[i][3])
                : "l"(ldA), "l"(ldB));
        }

#pragma unroll
        for (int ik = 0; ik < 4; ik++)
        {
#pragma unroll
            for (int im = 0; im < 8; im++)
            {
#pragma unroll
                for (int in = 0; in < 8; in++)
                {
                    asm volatile(
                        "mma.sync.aligned.m8n8k16.row.col.s32.s8.s8.s32 \
                        {%0, %1}, \
                        {%2}, {%3}, \
                        {%0, %1};"
                        : "+r"(frag_c[im * 8 + in][0]), "+r"(frag_c[im * 8 + in][1])
                        : "r"(frag_a[im][ik]), "r"(frag_b[in][ik])); // With an implicit __syncwarp() here.
                }
            }
        }
        __syncthreads();
    }

#pragma unroll
    for (int im = 0; im < 8; im++)
    {
#pragma unroll
        for (int in = 0; in < 8; in++)
        {
            auto idx = im * 8 + in;
            int32_t frag_d[2];
            *reinterpret_cast<int64_t *>(frag_d) = *reinterpret_cast<int64_t *>(&baseC[(im * 8 + laneId / 4) * ldc + in * 8 + (laneId & 3) * 2]); // I'm the reinterpret_cast master!
            frag_d[0] = frag_c[idx][0] * alpha + frag_d[0] * beta;
            frag_d[1] = frag_c[idx][1] * alpha + frag_d[1] * beta;
            *reinterpret_cast<int64_t *>(&baseC[(im * 8 + laneId / 4) * ldc + in * 8 + (laneId & 3) * 2]) = *reinterpret_cast<int64_t *>(frag_d);
        }
    }
}

void i8gemm(int M, int N, int K, int8_t *a, int8_t *b, int32_t *c, int32_t alpha, int32_t beta)
{
    dim3 threadsPerBlock(256);
    dim3 numBlocks((M + BLOCK_M - 1) / BLOCK_M, (N + BLOCK_N - 1) / BLOCK_N);
    i8gemm256x128x64<<<numBlocks, threadsPerBlock>>>(a, b, c, M, N, K, alpha, beta);
}